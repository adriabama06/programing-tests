#include "hip/hip_runtime.h"


#include <iostream>

struct ArrayObject
{
    int len;
    float* Array;
};

void printArray(ArrayObject* array)
{
    for(int i = 0; i < array->len; i++) {
        std::cout << "[" << i << "] = " << array->Array[i] << std::endl;
    }
    std::cout << std::endl;
}

__global__ void work(float* smallnumber, float* diferenceByPrecission, float* arr)
{
    int i = threadIdx.x;
    
    arr[i] = *smallnumber + (*diferenceByPrecission * (i+1));
}

struct ArrayObject* interpolate(float x, float y, int precission)
{
    float* arr;
    arr = (float*) calloc(precission, sizeof(float));

    float smallnumber = x;
    float diference = y - x;

    if(x > y) {
        smallnumber = y;
        diference = x - y;
    }

    float diferenceByPrecission = diference / (precission + 1);

    float* cuda_smallnumber;
    float* cuda_diferenceByPrecission;
    float* cuda_arr;

    hipMalloc(&cuda_smallnumber, sizeof(float));
    hipMalloc(&cuda_diferenceByPrecission, sizeof(float));
    hipMalloc(&cuda_arr, sizeof(float) * precission);

    hipMemcpy(cuda_smallnumber, &smallnumber, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cuda_diferenceByPrecission, &diferenceByPrecission, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cuda_arr, &arr, sizeof(float) * precission, hipMemcpyHostToDevice);

    work<<<1, precission>>>(cuda_smallnumber, cuda_diferenceByPrecission, cuda_arr);

    hipMemcpy(arr, cuda_arr, sizeof(float) * precission, hipMemcpyDeviceToHost);

    static struct ArrayObject toreturn = {0, 0};

    toreturn.len = precission;
    toreturn.Array = arr;
    
    return &toreturn;
}

int main()
{
    struct ArrayObject *result = interpolate(0, 100, 4);

    printArray(result);

    return 0;
}