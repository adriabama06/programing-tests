#include "hip/hip_runtime.h"


#include <iostream>

__global__ void vectorAdd(int* a, int* b, int* c) {
    int i = threadIdx.x;

    c[i] = a[i] + b[i];

    return;
}

void printArray(int *arr, int size) {
    for (int i = 0; i < size; i++) {
        std::cout << i << ": " << arr[i] << std::endl;
    }
    std::cout << std::endl;
    return;
}

int main()
{
    int a[] = {1, 2, 3};
    int b[] = {4, 5, 6};
    int c[sizeof(a) / sizeof(int)] = { 0 };

    // create pointers
    int* cuda_a = 0;
    int* cuda_b = 0;
    int* cuda_c = 0;

    // allocate memory on GPU
    hipMalloc(&cuda_a, sizeof(a)); // sizeof(a) is 12 (int (4), int (4), int (4)) in my computer
    hipMalloc(&cuda_b, sizeof(b));
    hipMalloc(&cuda_c, sizeof(c));

    // copy data to GPU
    hipMemcpy(cuda_a, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(b), hipMemcpyHostToDevice);
    hipMemcpy(cuda_c, c, sizeof(c), hipMemcpyHostToDevice);

    // call function on GPU
    // vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(cuda_a, cuda_b, cuda_c);
    vectorAdd<<<1, sizeof(a) / sizeof(int)>>>(cuda_a, cuda_b, cuda_c);
    
    // copy data back to CPU
    hipMemcpy(c, cuda_c, sizeof(c), hipMemcpyDeviceToHost);

    printArray(c, sizeof(c) / sizeof(int));

    return 0;
}